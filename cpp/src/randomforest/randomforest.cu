/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *	http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "../decisiontree/kernels/quantile.h"
#include "../decisiontree/memory.h"
#include "random/permute.h"
#include "random/rng.h"
#include "randomforest.h"
#include "score/scores.h"

namespace ML {

/**
 * @brief Construct RF_metrics.
 * @param[in] cfg_accuracy: accuracy.
 */
RF_metrics::RF_metrics(float cfg_accuracy)
  : rf_type(RF_type::CLASSIFICATION), accuracy(cfg_accuracy){};

/**
 * @brief Construct RF_metrics.
 * @param[in] cfg_mean_abs_error: mean absolute error.
 * @param[in] cfg_mean_squared_error: mean squared error.
 * @param[in] cfg_median_abs_error: median absolute error.
 */
RF_metrics::RF_metrics(double cfg_mean_abs_error, double cfg_mean_squared_error,
                       double cfg_median_abs_error)
  : rf_type(RF_type::REGRESSION),
    mean_abs_error(cfg_mean_abs_error),
    mean_squared_error(cfg_mean_squared_error),
    median_abs_error(cfg_median_abs_error){};

/**
 * @brief Print either accuracy metric for classification, or mean absolute error, mean squared error, 
   and median absolute error metrics for regression.
 */
void RF_metrics::print() {
  if (rf_type == RF_type::CLASSIFICATION) {
    std::cout << "Accuracy: " << accuracy << std::endl;
  } else if (rf_type == RF_type::REGRESSION) {
    std::cout << "Mean Absolute Error: " << mean_abs_error << std::endl;
    std::cout << "Mean Squared Error: " << mean_squared_error << std::endl;
    std::cout << "Median Absolute Error: " << median_abs_error << std::endl;
  }
}

/**
 * @brief Update labels so they are unique from 0 to n_unique_labels values.
		  Create/update an old label to new label map per random forest.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in,out] labels_map: map of old label values to new ones.
 * @param[in] verbose: debugging flag.
 */
void preprocess_labels(int n_rows, std::vector<int>& labels,
                       std::map<int, int>& labels_map, bool verbose) {
  std::pair<std::map<int, int>::iterator, bool> ret;
  int n_unique_labels = 0;

  if (verbose) std::cout << "Preprocessing labels\n";
  for (int i = 0; i < n_rows; i++) {
    ret = labels_map.insert(std::pair<int, int>(labels[i], n_unique_labels));
    if (ret.second) {
      n_unique_labels += 1;
    }
    if (verbose) std::cout << "Mapping " << labels[i] << " to ";
    labels[i] = ret.first->second;  //Update labels **IN-PLACE**
    if (verbose) std::cout << labels[i] << std::endl;
  }
  if (verbose) std::cout << "Finished preprocessing labels\n";
}

/**
 * @brief Revert label preprocessing effect, if needed.
 * @param[in] n_rows: number of rows (labels)
 * @param[in,out] labels: 1D labels array to be changed in-place.
 * @param[in] labels_map: map of old to new label values used during preprocessing.
 * @param[in] verbose: debugging flag.
 */
void postprocess_labels(int n_rows, std::vector<int>& labels,
                        std::map<int, int>& labels_map, bool verbose) {
  if (verbose) std::cout << "Postrocessing labels\n";
  std::map<int, int>::iterator it;
  int n_unique_cnt = labels_map.size();
  std::vector<int> reverse_map;
  reverse_map.resize(n_unique_cnt);
  for (auto it = labels_map.begin(); it != labels_map.end(); it++) {
    reverse_map[it->second] = it->first;
  }

  for (int i = 0; i < n_rows; i++) {
    if (verbose)
      std::cout << "Mapping " << labels[i] << " back to "
                << reverse_map[labels[i]] << std::endl;
    labels[i] = reverse_map[labels[i]];
  }
  if (verbose) std::cout << "Finished postrocessing labels\n";
}

/**
 * @brief Random forest hyper-parameter object default constructor (1 tree).
 */
RF_params::RF_params() : n_trees(1) {}

/**
 * @brief Random forest hyper-parameter object constructor to set n_trees member.
 */
RF_params::RF_params(int cfg_n_trees) : n_trees(cfg_n_trees) {}

/**
 * @brief Random forest hyper-parameter object constructor to set bootstrap, bootstrap_features, n_trees and rows_sample members.
 */
RF_params::RF_params(bool cfg_bootstrap, bool cfg_bootstrap_features,
                     int cfg_n_trees, float cfg_rows_sample)
  : bootstrap(cfg_bootstrap),
    bootstrap_features(cfg_bootstrap_features),
    n_trees(cfg_n_trees),
    rows_sample(cfg_rows_sample) {
  tree_params.bootstrap_features = cfg_bootstrap_features;
}

/**
 * @brief Random forest hyper-parameter object constructor to set all RF_params members.
 */
RF_params::RF_params(bool cfg_bootstrap, bool cfg_bootstrap_features,
                     int cfg_n_trees, float cfg_rows_sample,
                     DecisionTree::DecisionTreeParams cfg_tree_params)
  : bootstrap(cfg_bootstrap),
    bootstrap_features(cfg_bootstrap_features),
    n_trees(cfg_n_trees),
    rows_sample(cfg_rows_sample),
    tree_params(cfg_tree_params) {
  tree_params.bootstrap_features = cfg_bootstrap_features;
}

/**
 * @brief Check validity of all random forest hyper-parameters.
 */
void RF_params::validity_check() const {
  ASSERT((n_trees > 0), "Invalid n_trees %d", n_trees);
  ASSERT((rows_sample > 0) && (rows_sample <= 1.0),
         "rows_sample value %f outside permitted (0, 1] range", rows_sample);
  tree_params.validity_check();
}

/**
 * @brief Print all random forest hyper-parameters.
 */
void RF_params::print() const {
  std::cout << "bootstrap: " << bootstrap << std::endl;
  std::cout << "bootstrap features: " << bootstrap_features << std::endl;
  std::cout << "n_trees: " << n_trees << std::endl;
  std::cout << "rows_sample: " << rows_sample << std::endl;
  tree_params.print();
}

/**
 * @brief Construct rf (random forest) object.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] cfg_rf_params: Random forest hyper-parameter struct.
 * @param[in] cfg_rf_type: Random forest type. Only CLASSIFICATION is currently supported.
 */
template <typename T, typename L>
rf<T, L>::rf(RF_params cfg_rf_params, int cfg_rf_type)
  : rf_params(cfg_rf_params), rf_type(cfg_rf_type) {
  rf_params.validity_check();
}

/**
 * @brief Return number of trees in the forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 */
template <typename T, typename L>
int rf<T, L>::get_ntrees() {
  return rf_params.n_trees;
}

/**
 * @brief Print summary for all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 */
template <typename T, typename L>
void rf<T, L>::print_rf_summary() {
  const DecisionTree::DecisionTreeBase<T, L>* trees = get_trees_ptr();
  if (!trees) {
    std::cout << "Empty forest" << std::endl;
  } else {
    std::cout << "Forest has " << rf_params.n_trees << " trees, max_depth "
              << rf_params.tree_params.max_depth;
    std::cout << ", and max_leaves " << rf_params.tree_params.max_leaves
              << std::endl;
    for (int i = 0; i < rf_params.n_trees; i++) {
      std::cout << "Tree #" << i << std::endl;
      trees[i].print_tree_summary();
    }
  }
}

/**
 * @brief Print detailed view of all trees in the random forest.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 */
template <typename T, typename L>
void rf<T, L>::print_rf_detailed() {
  const DecisionTree::DecisionTreeBase<T, L>* trees = get_trees_ptr();
  if (!trees) {
    std::cout << "Empty forest" << std::endl;
  } else {
    std::cout << "Forest has " << rf_params.n_trees << " trees, max_depth "
              << rf_params.tree_params.max_depth;
    std::cout << ", and max_leaves " << rf_params.tree_params.max_leaves
              << std::endl;
    for (int i = 0; i < rf_params.n_trees; i++) {
      std::cout << "Tree #" << i << std::endl;
      trees[i].print();
    }
  }
}

/**
 * @brief Sample row IDs for tree fitting and bootstrap if requested.
 * @tparam T: data type for input data (float or double).
 * @tparam L: data type for labels (int type for classification, T type for regression).
 * @param[in] handle: cumlHandle
 * @param[in] tree_id: unique tree ID
 * @param[in] n_rows: total number of data samples.
 * @param[in] n_sampled_rows: number of rows used for training
 * @param[in, out] selected_rows: already allocated array w/ row IDs
 * @param[in, out] sorted_selected_rows: already allocated array. Will contain sorted row IDs.
 * @param[in, out] rows_temp_storage: temp. storage used for sorting (previously allocated).
 * @param[in] temp_storage_bytes: size in bytes of rows_temp_storage.
 */
template <typename T, typename L>
void rf<T, L>::prepare_fit_per_tree(const ML::cumlHandle_impl& handle,
                                    int tree_id, int n_rows, int n_sampled_rows,
                                    unsigned int* selected_rows,
                                    unsigned int* sorted_selected_rows,
                                    char* rows_temp_storage,
                                    size_t temp_storage_bytes) {
  hipStream_t stream = handle.getStream();

  if (rf_params.bootstrap) {
    MLCommon::Random::Rng r(
      tree_id *
      1000);  // Ensure the seed for each tree is different and meaningful.
    r.uniformInt(selected_rows, n_sampled_rows, (unsigned int)0,
                 (unsigned int)n_rows, stream);
    //thrust::sequence(thrust::cuda::par.on(stream), sorted_selected_rows,
    //           sorted_selected_rows + n_sampled_rows);

    CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
      (void*)rows_temp_storage, temp_storage_bytes, selected_rows,
      sorted_selected_rows, n_sampled_rows, 0, 8 * sizeof(unsigned int),
      stream));
  } else {  // Sampling w/o replacement
    MLCommon::device_buffer<unsigned int>* inkeys =
      new MLCommon::device_buffer<unsigned int>(handle.getDeviceAllocator(),
                                                stream, n_rows);
    MLCommon::device_buffer<unsigned int>* outkeys =
      new MLCommon::device_buffer<unsigned int>(handle.getDeviceAllocator(),
                                                stream, n_rows);
    thrust::sequence(thrust::cuda::par.on(stream), inkeys->data(),
                     inkeys->data() + n_rows);
    int* perms = nullptr;
    MLCommon::Random::permute(perms, outkeys->data(), inkeys->data(), 1, n_rows,
                              false, stream);
    // outkeys has more rows than selected_rows; doing the shuffling before the resize to differentiate the per-tree rows sample.
    CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
      (void*)rows_temp_storage, temp_storage_bytes, outkeys->data(),
      sorted_selected_rows, n_sampled_rows, 0, 8 * sizeof(unsigned int),
      stream));
    inkeys->release(stream);
    outkeys->release(stream);
    delete inkeys;
    delete outkeys;
  }
}

template <typename T, typename L>
void rf<T, L>::error_checking(const T* input, L* predictions, int n_rows,
                              int n_cols, bool predict) const {
  if (predict) {
    ASSERT(get_trees_ptr(), "Cannot predict! No trees in the forest.");
    ASSERT(predictions != nullptr,
           "Error! User has not allocated memory for predictions.");
  } else {
    ASSERT(!get_trees_ptr(), "Cannot fit an existing forest.");
  }
  ASSERT((n_rows > 0), "Invalid n_rows %d", n_rows);
  ASSERT((n_cols > 0), "Invalid n_cols %d", n_cols);

  bool input_is_dev_ptr = is_dev_ptr(input);
  bool preds_is_dev_ptr = is_dev_ptr(predictions);

  if (!input_is_dev_ptr || (input_is_dev_ptr != preds_is_dev_ptr)) {
    ASSERT(false,
           "RF Error: Expected both input and labels/predictions to be GPU "
           "pointers");
  }
}

/**
 * @brief Construct rfClassifier object.
 * @tparam T: data type for input data (float or double).
 * @param[in] cfg_rf_params: Random forest hyper-parameter struct.
 */
template <typename T>
rfClassifier<T>::rfClassifier(RF_params cfg_rf_params)
  : rf<T, int>::rf(cfg_rf_params, RF_type::CLASSIFICATION){};

/**
 * @brief Destructor for random forest classifier object.
 * @tparam T: data type for input data (float or double).
 */
template <typename T>
rfClassifier<T>::~rfClassifier() {
  delete[] trees;
}

/**
 * @brief Return a const pointer to decision trees.
 * @tparam T: data type for input data (float or double).
 */
template <typename T>
const DecisionTree::DecisionTreeClassifier<T>* rfClassifier<T>::get_trees_ptr()
  const {
  return trees;
}

/**
 * @brief Build (i.e., fit, train) random forest classifier for input data.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (int only), with one label per training sample. Device pointer.
				  Assumption: labels were preprocessed to map to ascending numbers from 0;
				  needed for current gini impl in decision tree
 * @param[in] n_unique_labels: #unique label values (known during preprocessing)
 */
template <typename T>
void rfClassifier<T>::fit(const cumlHandle& user_handle, T* input, int n_rows,
                          int n_cols, int* labels, int n_unique_labels) {
  this->error_checking(input, labels, n_rows, n_cols, false);

  trees = new DecisionTree::DecisionTreeClassifier<T>[this->rf_params.n_trees];

  int n_sampled_rows = this->rf_params.rows_sample * n_rows;

  const cumlHandle_impl& handle = user_handle.getImpl();
  hipStream_t stream = user_handle.getStream();

  // Select n_sampled_rows (with replacement) numbers from [0, n_rows) per tree.
  // selected_rows: randomly generated IDs for bootstrapped samples (w/ replacement); a device ptr.
  MLCommon::device_buffer<unsigned int> selected_rows(
    handle.getDeviceAllocator(), stream, n_sampled_rows);
  MLCommon::device_buffer<unsigned int> sorted_selected_rows(
    handle.getDeviceAllocator(), stream, n_sampled_rows);

  // Will sort selected_rows (row IDs), prior to fit, to improve access patterns
  MLCommon::device_buffer<char>* rows_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
    rows_temp_storage, temp_storage_bytes, selected_rows.data(),
    sorted_selected_rows.data(), n_sampled_rows, 0, 8 * sizeof(unsigned int),
    stream));
  // Allocate temporary storage
  rows_temp_storage = new MLCommon::device_buffer<char>(
    handle.getDeviceAllocator(), stream, temp_storage_bytes);
  std::shared_ptr<TemporaryMemory<T, int>> tempmem =
    std::make_shared<TemporaryMemory<T, int>>(
      user_handle.getImpl(), n_sampled_rows, n_cols, 1, n_unique_labels,
      this->rf_params.tree_params.n_bins,
      this->rf_params.tree_params.split_algo);
  if ((this->rf_params.tree_params.split_algo == SPLIT_ALGO::GLOBAL_QUANTILE) &&
      !(this->rf_params.tree_params.quantile_per_tree)) {
    preprocess_quantile(input, nullptr, n_sampled_rows, n_cols, n_rows,
                        this->rf_params.tree_params.n_bins, tempmem);
  }
  for (int i = 0; i < this->rf_params.n_trees; i++) {
    this->prepare_fit_per_tree(handle, i, n_rows, n_sampled_rows,
                               selected_rows.data(),
                               sorted_selected_rows.data(),
                               rows_temp_storage->data(), temp_storage_bytes);

    /* Build individual tree in the forest.
		   - input is a pointer to orig data that have n_cols features and n_rows rows.
		   - n_sampled_rows: # rows sampled for tree's bootstrap sample.
		   - sorted_selected_rows: points to a list of row #s (w/ n_sampled_rows elements) used to build the bootstrapped sample.
		   Expectation: Each tree node will contain (a) # n_sampled_rows and (b) a pointer to a list of row numbers w.r.t original data.
		*/

    trees[i].fit(user_handle, input, n_cols, n_rows, labels,
                 sorted_selected_rows.data(), n_sampled_rows, n_unique_labels,
                 this->rf_params.tree_params, tempmem);
  }

  //Cleanup
  rows_temp_storage->release(stream);
  selected_rows.release(stream);
  sorted_selected_rows.release(stream);
  tempmem.reset();
  delete rows_temp_storage;
}

/**
 * @brief Predict target feature for input data; n-ary classification for single feature supported.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
void rfClassifier<T>::predict(const cumlHandle& user_handle, const T* input,
                              int n_rows, int n_cols, int* predictions,
                              bool verbose) const {
  this->error_checking(input, predictions, n_rows, n_cols, true);

  std::vector<int> h_predictions(n_rows);
  const cumlHandle_impl& handle = user_handle.getImpl();
  hipStream_t stream = user_handle.getStream();

  std::vector<T> h_input(n_rows * n_cols);
  MLCommon::updateHost(h_input.data(), input, n_rows * n_cols, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  int row_size = n_cols;

  for (int row_id = 0; row_id < n_rows; row_id++) {
    if (verbose) {
      std::cout << "\n\n";
      std::cout << "Predict for sample: ";
      for (int i = 0; i < n_cols; i++)
        std::cout << h_input[row_id * row_size + i] << ", ";
      std::cout << std::endl;
    }

    std::map<int, int> prediction_to_cnt;
    std::pair<std::map<int, int>::iterator, bool> ret;
    int max_cnt_so_far = 0;
    int majority_prediction = -1;

    for (int i = 0; i < this->rf_params.n_trees; i++) {
      //Return prediction for one sample.
      if (verbose) {
        std::cout << "Printing tree " << i << std::endl;
        trees[i].print();
      }
      int prediction;
      trees[i].predict(user_handle, &h_input[row_id * row_size], 1, n_cols,
                       &prediction, verbose);
      ret = prediction_to_cnt.insert(std::pair<int, int>(prediction, 1));
      if (!(ret.second)) {
        ret.first->second += 1;
      }
      if (max_cnt_so_far < ret.first->second) {
        max_cnt_so_far = ret.first->second;
        majority_prediction = ret.first->first;
      }
    }

    h_predictions[row_id] = majority_prediction;
  }

  MLCommon::updateDevice(predictions, h_predictions.data(), n_rows, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

/**
 * @brief Predict target feature for input data; n-ary classification for single feature supported.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
void rfClassifier<T>::predictAllDTs(const cumlHandle& user_handle,
                                    const T* input, int n_rows, int n_cols,
                                    int* predictions, bool verbose) const {
  ASSERT(this->trees, "Cannot predict! No trees in the forest.");
  ASSERT((n_rows > 0), "Invalid n_rows %d", n_rows);
  ASSERT((n_cols > 0), "Invalid n_cols %d", n_cols);
  ASSERT(predictions != nullptr,
         "Error! User has not allocated memory for predictions.");
  int row_size = n_cols;

  int pred_id = 0;
  for (int row_id = 0; row_id < n_rows; row_id++) {
    if (verbose) {
      std::cout << "\n\n";
      std::cout << "Predict for sample: ";
      for (int i = 0; i < n_cols; i++)
        std::cout << input[row_id * row_size + i] << ", ";
      std::cout << std::endl;
    }

    std::map<int, int> prediction_to_cnt;
    std::pair<std::map<int, int>::iterator, bool> ret;
    int max_cnt_so_far = 0;
    int majority_prediction = -1;

    for (int i = 0; i < this->rf_params.n_trees; i++) {
      //Return prediction for one sample.
      if (verbose) {
        std::cout << "Printing tree " << i << std::endl;
        //this->trees[i].print();
      }
      int prediction;
      this->trees[i].predict(user_handle, &input[row_id * row_size], 1, n_cols,
                             &prediction, verbose);

      predictions[pred_id] = prediction;
      pred_id++;
    }
  }
}

/**
 * @brief Predict target feature for input data and validate against ref_labels.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
void rfClassifier<T>::predictAllDTs(const cumlHandle& user_handle,
                                    const T* input, int n_rows, int n_cols,
                                    int* predictions, bool verbose) const {
  ASSERT(this->trees, "Cannot predict! No trees in the forest.");
  ASSERT((n_rows > 0), "Invalid n_rows %d", n_rows);
  ASSERT((n_cols > 0), "Invalid n_cols %d", n_cols);
  ASSERT(predictions != nullptr,
         "Error! User has not allocated memory for predictions.");
  int row_size = n_cols;

  int pred_id = 0;
  for (int row_id = 0; row_id < n_rows; row_id++) {
    if (verbose) {
      std::cout << "\n\n";
      std::cout << "Predict for sample: ";
      for (int i = 0; i < n_cols; i++)
        std::cout << input[row_id * row_size + i] << ", ";
      std::cout << std::endl;
    }

    std::map<int, int> prediction_to_cnt;
    std::pair<std::map<int, int>::iterator, bool> ret;
    int max_cnt_so_far = 0;
    int majority_prediction = -1;

    for (int i = 0; i < this->rf_params.n_trees; i++) {
      //Return prediction for one sample.
      if (verbose) {
        std::cout << "Printing tree " << i << std::endl;
        //this->trees[i].print();
      }
      int prediction;
      this->trees[i].predict(user_handle, &input[row_id * row_size], 1, n_cols,
                             &prediction, verbose);

      predictions[pred_id] = prediction;
      pred_id++;
    }
  }
}

/**
 * @brief Predict target feature for input data and validate against ref_labels.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
RF_metrics rfClassifier<T>::score(const cumlHandle& user_handle, const T* input,
                                  const int* ref_labels, int n_rows, int n_cols,
                                  int* predictions, bool verbose) const {
  predict(user_handle, input, n_rows, n_cols, predictions, verbose);

  hipStream_t stream = user_handle.getImpl().getStream();
  auto d_alloc = user_handle.getDeviceAllocator();
  float accuracy = MLCommon::Score::accuracy_score(predictions, ref_labels,
                                                   n_rows, d_alloc, stream);
  RF_metrics stats(accuracy);
  if (verbose) stats.print();

  /* TODO: Potentially augment RF_metrics w/ more metrics (e.g., precision, F1, etc.).
	   For non binary classification problems (i.e., one target and  > 2 labels), need avg for each of these metrics */
  return stats;
}

/**
 * @brief Construct rfRegressor object.
 * @tparam T: data type for input data (float or double).
 * @param[in] cfg_rf_params: Random forest hyper-parameter struct.
 */
template <typename T>
rfRegressor<T>::rfRegressor(RF_params cfg_rf_params)
  : rf<T, T>::rf(cfg_rf_params, RF_type::REGRESSION){};

/**
 * @brief Destructor for random forest regressor object.
 * @tparam T: data type for input data (float or double).
 */
template <typename T>
rfRegressor<T>::~rfRegressor() {
  delete[] trees;
}

/**
 * @brief Return a const pointer to decision trees.
 * @tparam T: data type for input data (float or double).
 */
template <typename T>
const DecisionTree::DecisionTreeRegressor<T>* rfRegressor<T>::get_trees_ptr()
  const {
  return trees;
}

/**
 * @brief Build (i.e., fit, train) random forest regressor for input data.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (float or double), with one label per training sample. Device pointer.
 */
template <typename T>
void rfRegressor<T>::fit(const cumlHandle& user_handle, T* input, int n_rows,
                         int n_cols, T* labels) {
  this->error_checking(input, labels, n_rows, n_cols, false);

  trees = new DecisionTree::DecisionTreeRegressor<T>[this->rf_params.n_trees];

  int n_sampled_rows = this->rf_params.rows_sample * n_rows;

  const cumlHandle_impl& handle = user_handle.getImpl();
  hipStream_t stream = user_handle.getStream();

  // Select n_sampled_rows (with replacement) numbers from [0, n_rows) per tree.
  // selected_rows: randomly generated IDs for bootstrapped samples (w/ replacement); a device ptr.
  MLCommon::device_buffer<unsigned int> selected_rows(
    handle.getDeviceAllocator(), stream, n_sampled_rows);
  MLCommon::device_buffer<unsigned int> sorted_selected_rows(
    handle.getDeviceAllocator(), stream, n_sampled_rows);

  // Will sort selected_rows (row IDs), prior to fit, to improve access patterns
  MLCommon::device_buffer<char>* rows_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  CUDA_CHECK(hipcub::DeviceRadixSort::SortKeys(
    rows_temp_storage, temp_storage_bytes, selected_rows.data(),
    sorted_selected_rows.data(), n_sampled_rows, 0, 8 * sizeof(unsigned int),
    stream));
  // Allocate temporary storage
  rows_temp_storage = new MLCommon::device_buffer<char>(
    handle.getDeviceAllocator(), stream, temp_storage_bytes);
  std::shared_ptr<TemporaryMemory<T, T>> tempmem =
    std::make_shared<TemporaryMemory<T, T>>(
      user_handle.getImpl(), n_sampled_rows, n_cols, 1, 1,
      this->rf_params.tree_params.n_bins,
      this->rf_params.tree_params.split_algo);

  if ((this->rf_params.tree_params.split_algo == SPLIT_ALGO::GLOBAL_QUANTILE) &&
      !(this->rf_params.tree_params.quantile_per_tree)) {
    preprocess_quantile(input, nullptr, n_sampled_rows, n_cols, n_rows,
                        this->rf_params.tree_params.n_bins, tempmem);
  }
  for (int i = 0; i < this->rf_params.n_trees; i++) {
    this->prepare_fit_per_tree(handle, i, n_rows, n_sampled_rows,
                               selected_rows.data(),
                               sorted_selected_rows.data(),
                               rows_temp_storage->data(), temp_storage_bytes);

    /* Build individual tree in the forest.
		   - input is a pointer to orig data that have n_cols features and n_rows rows.
		   - n_sampled_rows: # rows sampled for tree's bootstrap sample.
		   - sorted_selected_rows: points to a list of row #s (w/ n_sampled_rows elements) used to build the bootstrapped sample.
		   Expectation: Each tree node will contain (a) # n_sampled_rows and (b) a pointer to a list of row numbers w.r.t original data.
		*/

    trees[i].fit(user_handle, input, n_cols, n_rows, labels,
                 sorted_selected_rows.data(), n_sampled_rows,
                 this->rf_params.tree_params, tempmem);
  }
  //Cleanup
  rows_temp_storage->release(stream);
  selected_rows.release(stream);
  sorted_selected_rows.release(stream);
  tempmem.reset();
  delete rows_temp_storage;
}

/**
 * @brief Predict target feature for input data; regression for single feature supported.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
void rfRegressor<T>::predict(const cumlHandle& user_handle, const T* input,
                             int n_rows, int n_cols, T* predictions,
                             bool verbose) const {
  this->error_checking(input, predictions, n_rows, n_cols, true);

  std::vector<T> h_predictions(n_rows);
  const cumlHandle_impl& handle = user_handle.getImpl();
  hipStream_t stream = user_handle.getStream();

  std::vector<T> h_input(n_rows * n_cols);
  MLCommon::updateHost(h_input.data(), input, n_rows * n_cols, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));

  int row_size = n_cols;

  for (int row_id = 0; row_id < n_rows; row_id++) {
    if (verbose) {
      std::cout << "\n\n";
      std::cout << "Predict for sample: ";
      for (int i = 0; i < n_cols; i++)
        std::cout << h_input[row_id * row_size + i] << ", ";
      std::cout << std::endl;
    }

    T sum_predictions = 0;

    for (int i = 0; i < this->rf_params.n_trees; i++) {
      //Return prediction for one sample.
      if (verbose) {
        std::cout << "Printing tree " << i << std::endl;
        trees[i].print();
      }
      T prediction;
      trees[i].predict(user_handle, &h_input[row_id * row_size], 1, n_cols,
                       &prediction, verbose);
      sum_predictions += prediction;
    }
    // Random forest's prediction is the arithmetic mean of all its decision tree predictions.
    h_predictions[row_id] = sum_predictions / this->rf_params.n_trees;
  }

  MLCommon::updateDevice(predictions, h_predictions.data(), n_rows, stream);
  CUDA_CHECK(hipStreamSynchronize(stream));
}

/**
 * @brief Predict target feature for input data and validate against ref_labels.
 * @tparam T: data type for input data (float or double).
 * @param[in] user_handle: cumlHandle.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
template <typename T>
RF_metrics rfRegressor<T>::score(const cumlHandle& user_handle, const T* input,
                                 const T* ref_labels, int n_rows, int n_cols,
                                 T* predictions, bool verbose) const {
  predict(user_handle, input, n_rows, n_cols, predictions, verbose);

  hipStream_t stream = user_handle.getImpl().getStream();
  auto d_alloc = user_handle.getDeviceAllocator();

  double mean_abs_error, mean_squared_error, median_abs_error;
  MLCommon::Score::regression_metrics(predictions, ref_labels, n_rows, d_alloc,
                                      stream, mean_abs_error,
                                      mean_squared_error, median_abs_error);
  RF_metrics stats(mean_abs_error, mean_squared_error, median_abs_error);
  if (verbose) stats.print();

  return stats;
}

template class rf<float, int>;
template class rf<float, float>;
template class rf<double, int>;
template class rf<double, double>;

template class rfClassifier<float>;
template class rfClassifier<double>;

template class rfRegressor<float>;
template class rfRegressor<double>;

// Stateless API functions: fit, predict and score

// ----------------------------- Classification ----------------------------------- //

/**
 * @brief Build (i.e., fit, train) random forest classifier for input data of type float.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] rf_classifier: pointer to the rfClassifier object, previously constructed by the user.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (int only), with one label per training sample. Device pointer.
				  Assumption: labels were preprocessed to map to ascending numbers from 0;
				  needed for current gini impl. in decision tree
 * @param[in] n_unique_labels: #unique label values (known during preprocessing)
 */
void fit(const cumlHandle& user_handle, rfClassifier<float>* rf_classifier,
         float* input, int n_rows, int n_cols, int* labels,
         int n_unique_labels) {
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels);
}

/**
 * @brief Build (i.e., fit, train) random forest classifier for input data of type double.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] rf_classifier: pointer to the rfClassifier object, previously constructed by the user.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (int only), with one label per training sample. Device pointer.
				  Assumption: labels were preprocessed to map to ascending numbers from 0;
				  needed for current gini impl. in decision tree
 * @param[in] n_unique_labels: #unique label values (known during preprocessing)
 */
void fit(const cumlHandle& user_handle, rfClassifier<double>* rf_classifier,
         double* input, int n_rows, int n_cols, int* labels,
         int n_unique_labels) {
  rf_classifier->fit(user_handle, input, n_rows, n_cols, labels,
                     n_unique_labels);
}

/**
 * @brief Predict target feature for input data of type float; n-ary classification for single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predict(const cumlHandle& user_handle,
             const rfClassifier<float>* rf_classifier, const float* input,
             int n_rows, int n_cols, int* predictions, bool verbose) {
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         verbose);
}

/**
 * @brief Predict target feature for input data of type double; n-ary classification for single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predict(const cumlHandle& user_handle,
             const rfClassifier<double>* rf_classifier, const double* input,
             int n_rows, int n_cols, int* predictions, bool verbose) {
  rf_classifier->predict(user_handle, input, n_rows, n_cols, predictions,
                         verbose);
}

/**
 * @brief Predict target feature for input data of type float; n-ary classification for single feature supported.
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predictAllDTs(const cumlHandle& user_handle,
                   const rfClassifier<float>* rf_classifier, const float* input,
                   int n_rows, int n_cols, int* predictions, bool verbose) {
  rf_classifier->predictAllDTs(user_handle, input, n_rows, n_cols, predictions,
                               verbose);
}

/**
 * @brief Predict target feature for input data of type double; n-ary classification for single feature supported.
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predictAllDTs(const cumlHandle& user_handle,
                   const rfClassifier<double>* rf_classifier,
                   const double* input, int n_rows, int n_cols,
                   int* predictions, bool verbose) {
  rf_classifier->predictAllDTs(user_handle, input, n_rows, n_cols, predictions,
                               verbose);
}

/**
 * @brief Predict target feature for input data of type float and validate against ref_labels.
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predictAllDTs(const cumlHandle& user_handle,
                   const rfClassifier<float>* rf_classifier, const float* input,
                   int n_rows, int n_cols, int* predictions, bool verbose) {
  rf_classifier->predictAllDTs(user_handle, input, n_rows, n_cols, predictions,
                               verbose);
}

/**
 * @brief Predict target feature for input data of type double; n-ary classification for single feature supported.
 * @param[in] user_handle: cumlHandle (currently unused; API placeholder)
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. CPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. CPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predictAllDTs(const cumlHandle& user_handle,
                   const rfClassifier<double>* rf_classifier,
                   const double* input, int n_rows, int n_cols,
                   int* predictions, bool verbose) {
  rf_classifier->predictAllDTs(user_handle, input, n_rows, n_cols, predictions,
                               verbose);
}

/**
 * @brief Predict target feature for input data of type float and validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
RF_metrics score(const cumlHandle& user_handle,
                 const rfClassifier<float>* rf_classifier, const float* input,
                 const int* ref_labels, int n_rows, int n_cols,
                 int* predictions, bool verbose) {
  return rf_classifier->score(user_handle, input, ref_labels, n_rows, n_cols,
                              predictions, verbose);
}

/**
 * @brief Predict target feature for input data of type double and validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_classifier: pointer to the rfClassifier object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
RF_metrics score(const cumlHandle& user_handle,
                 const rfClassifier<double>* rf_classifier, const double* input,
                 const int* ref_labels, int n_rows, int n_cols,
                 int* predictions, bool verbose) {
  return rf_classifier->score(user_handle, input, ref_labels, n_rows, n_cols,
                              predictions, verbose);
}

RF_params set_rf_class_obj(int max_depth, int max_leaves, float max_features,
                           int n_bins, int split_algo, int min_rows_per_node,
                           bool bootstrap_features, bool bootstrap, int n_trees,
                           float rows_sample, CRITERION split_criterion,
                           bool quantile_per_tree) {
  DecisionTree::DecisionTreeParams tree_params(
    max_depth, max_leaves, max_features, n_bins, split_algo, min_rows_per_node,
    bootstrap_features, split_criterion, quantile_per_tree);
  RF_params rf_params(bootstrap, bootstrap_features, n_trees, rows_sample,
                      tree_params);
  return rf_params;
}

// ----------------------------- Regression ----------------------------------- //

/**
 * @brief Build (i.e., fit, train) random forest regressor for input data of type float.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] rf_regreesor: pointer to the rfRegressor object, previously constructed by the user.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (float), with one label per training sample. Device pointer.
 */
void fit(const cumlHandle& user_handle, rfRegressor<float>* rf_regressor,
         float* input, int n_rows, int n_cols, float* labels) {
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels);
}

/**
 * @brief Build (i.e., fit, train) random forest regressor for input data of type double.
 * @param[in] user_handle: cumlHandle
 * @param[in,out] rf_regressor: pointer to the rfRegressor object, previously constructed by the user.
 * @param[in] input: train data (n_rows samples, n_cols features) in column major format, excluding labels. Device pointer.
 * @param[in] n_rows: number of training data samples.
 * @param[in] n_cols: number of features (i.e., columns) excluding target feature.
 * @param[in] labels: 1D array of target features (double), with one label per training sample. Device pointer.
 */
void fit(const cumlHandle& user_handle, rfRegressor<double>* rf_regressor,
         double* input, int n_rows, int n_cols, double* labels) {
  rf_regressor->fit(user_handle, input, n_rows, n_cols, labels);
}

/**
 * @brief Predict target feature for input data of type float; regression for single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_regressor: pointer to the rfRegressor object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predict(const cumlHandle& user_handle,
             const rfRegressor<float>* rf_regressor, const float* input,
             int n_rows, int n_cols, float* predictions, bool verbose) {
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions,
                        verbose);
}

/**
 * @brief Predict target feature for input data of type double; regression for single feature supported.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_regressor: pointer to the rfRegressor object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
void predict(const cumlHandle& user_handle,
             const rfRegressor<double>* rf_regressor, const double* input,
             int n_rows, int n_cols, double* predictions, bool verbose) {
  rf_regressor->predict(user_handle, input, n_rows, n_cols, predictions,
                        verbose);
}

/**
 * @brief Predict target feature for input data of type float and validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_regressor: pointer to the rfRegressor object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
RF_metrics score(const cumlHandle& user_handle,
                 const rfRegressor<float>* rf_regressor, const float* input,
                 const float* ref_labels, int n_rows, int n_cols,
                 float* predictions, bool verbose) {
  return rf_regressor->score(user_handle, input, ref_labels, n_rows, n_cols,
                             predictions, verbose);
}

/**
 * @brief Predict target feature for input data of type double and validate against ref_labels.
 * @param[in] user_handle: cumlHandle.
 * @param[in] rf_regressor: pointer to the rfRegressor object. The user should have previously called fit to build the random forest.
 * @param[in] input: test data (n_rows samples, n_cols features) in row major format. GPU pointer.
 * @param[in] ref_labels: label values for cross validation (n_rows elements); GPU pointer.
 * @param[in] n_rows: number of  data samples.
 * @param[in] n_cols: number of features (excluding target feature).
 * @param[in, out] predictions: n_rows predicted labels. GPU pointer, user allocated.
 * @param[in] verbose: flag for debugging purposes.
 */
RF_metrics score(const cumlHandle& user_handle,
                 const rfRegressor<double>* rf_regressor, const double* input,
                 const double* ref_labels, int n_rows, int n_cols,
                 double* predictions, bool verbose) {
  return rf_regressor->score(user_handle, input, ref_labels, n_rows, n_cols,
                             predictions, verbose);
}

};  // namespace ML
// end namespace ML
